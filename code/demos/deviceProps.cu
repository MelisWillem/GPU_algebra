#include"vecUtils.h"

int main(){
    hipDeviceProp_t prop;

    // First find out how many devices there are.
    int count;
    HANDLE_ERROR( hipGetDeviceCount(&count) );
    for(int i=0; i < count; ++i){
        HANDLE_ERROR( hipGetDeviceProperties(&prop, i ) );

        std::cout << "found device with name=" << prop.name << std::endl;
    }
}
